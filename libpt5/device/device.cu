#include "hip/hip_runtime.h"
#include <optix_device.h>
#include <stdint.h>

#include "../vector_math.h"
#include "../LaunchParams.hpp"
#include "../sbt.hpp"
#include "../material.h"
#include "intersection.cuh"
#include "util.cuh"
#include "math.cuh"

namespace pt5{

extern "C" __constant__ LaunchParams launchParams;


enum {
	SURFACE_RAY_TYPE=0,
	RAY_TYPE_COUNT
};


struct PaylaodData{
	float3 emission;
	float3 albedo;
	RNG rng;
	float pContinue;
	float3 ray_o;
	float3 ray_d;
};


extern "C" __global__ void __closesthit__radiance(){
	PaylaodData& payload = *(PaylaodData*)getPRD<PaylaodData>();
	const HitgroupSBTData& sbtData = *(HitgroupSBTData*)optixGetSbtDataPointer();

	const int primID = optixGetPrimitiveIndex();
	const Intersection is = make_intersection(sbtData, primID);


	float3 tangent;
	float3 binromal;
	if(fabs(is.n.x) > fabs(is.n.z)){
		binromal = make_float3(-is.n.y, is.n.x, 0);
	}
	else{
		binromal = make_float3(0, -is.n.z, is.n.y);
	}

	binromal = normalize(binromal);
	tangent = cross(binromal, is.n);

	float3 ray_d = sample_cosine_hemisphere(payload.rng.uniform(), payload.rng.uniform());

	const Material mtl_default;
	const Material& mtl = (is.material)? *is.material : mtl_default;

	payload.pContinue = max(mtl.albedo.x, max(mtl.albedo.y, mtl.albedo.z));
	payload.emission = mtl.emission;
	payload.albedo = (mtl.texture>0)?
		make_float3(tex2D<float4>(mtl.texture, is.uv.x, is.uv.y))
		: mtl.albedo;
	payload.ray_o = is.p + 0.001*is.ng;
	payload.ray_d = ray_d.x*tangent + ray_d.y*binromal + ray_d.z*is.n;
}


extern "C" __global__ void __miss__radiance(){
	MissSBTData& sbtData = *(MissSBTData*)optixGetSbtDataPointer();
	PaylaodData& payload = *(PaylaodData*)getPRD<PaylaodData>();
	payload.emission = sbtData.background;
	payload.albedo = make_float3(0);
	payload.pContinue = 0;
}


extern "C" __global__ void __raygen__render(){
	RaygenSBTData& sbtData = *(RaygenSBTData*)optixGetSbtDataPointer();

	const int ix = optixGetLaunchIndex().x;
	const int iy = optixGetLaunchIndex().y;
	const uint2 frameSize = launchParams.image.size;
	const int pixel_index = iy*frameSize.x + ix;

	PaylaodData payload;
	uint32_t u0, u1;
	packPointer(&payload, u0, u1);

	float3 accum = make_float3(0,0,0);
	payload.rng = RNG(pixel_index);

	for(int i=0; i<launchParams.spp; i++){

		const float x =  (2*(ix+payload.rng.uniform()) - frameSize.x)/frameSize.x;
		const float y = -(2*(iy+payload.rng.uniform()) - frameSize.y)/frameSize.x;

		payload.pContinue = 1;
		payload.ray_o = launchParams.camera.position;
		payload.ray_d = launchParams.camera.view(x,y);

		float3 throuput = make_float3(1);

		while(payload.rng.uniform() < payload.pContinue){

			throuput /= payload.pContinue;

			const float tmax = 1e20;
			const float tmin = 0;

			optixTrace(
				sbtData.traversable,
				payload.ray_o,
				payload.ray_d,
				tmin, tmax, 0,
				OptixVisibilityMask(255),
				OPTIX_RAY_FLAG_DISABLE_ANYHIT,
				SURFACE_RAY_TYPE,
				RAY_TYPE_COUNT,
				SURFACE_RAY_TYPE,
				u0, u1);

			accum += throuput * payload.emission;
			throuput *= payload.albedo;
		}
	}

	launchParams.image.pixels[pixel_index] = make_float4(accum/launchParams.spp, 1);
}

} // pt5 namespace